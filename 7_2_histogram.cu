#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// 2.386ms
template <int INNER_LOOP_SIZE>
__global__ void histgram(int *hist_data, int *bin_data)
{
    // 使用共享内存
    __shared__ int bin[256];

    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // 初始化共享内存中的bin数组

    {
        bin[tid] = 0;
    }
    __syncthreads();

    // 计算直方图
    for (int i = 0; i < INNER_LOOP_SIZE; i++)
    {
        atomicAdd(&bin[hist_data[gtid * INNER_LOOP_SIZE + i]], 1);
    }
    __syncthreads();

    // 将共享内存中的bin数组累加到全局内存中的bin_data数组

    {
        atomicAdd(&bin_data[tid], bin[tid]);
    }
    // atomicAdd(&bin_data[hist_data[gtid]], 1);
}

bool CheckResult(int *out, int *groudtruth, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (out[i] != groudtruth[i])
        {
            return false;
        }
    }
    return true;
}

int main()
{
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));
    ;
    for (int j = 0; j < 256; j++)
    {
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    const int innerLoopSize = 200;
    int GridSize = std::min((N + 256 - 1) / (256 * innerLoopSize), deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<innerLoopSize><<<Grid, Block>>>(hist_data, bin_data);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(bin, groudtruth, 256);
    if (is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < 256; i++)
        {
            printf("%lf ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram latency = %f ms\n", milliseconds);

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}